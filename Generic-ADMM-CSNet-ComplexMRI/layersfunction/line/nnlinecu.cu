#include "hip/hip_runtime.h"
#include "mex.h"
#include "gpu/mxGPUArray.h"
#include "math.h"

#define IDX2(X, n1, n2, i1, i2) (X[(i2)*(n1) + (i1)])
#define IDX3(X, n1, n2, n3, i1, i2, i3) (X[(i3)*((n1)*(n2)) + (i2)*(n1) + (i1)])
#define IDX4(X, n1, n2, n3, n4, i1, i2, i3, i4) (X[(i4)*((n1)*(n2)*(n3)) + (i3)*((n1)*(n2)) + (i2)*(n1) + (i1)])

/* nnline_ker evaluate xvar with non-linear functions with control points */
void __global__ nnline_ker(
        const float *xlab, const float *ylab, const float *xvar,
        float *yvar, int M, int N, int D, int P)
{
    int k, l, p, q;
    
    int total_number = M * N * D;
    int n = blockDim.x * blockIdx.x + threadIdx.x;
    int total_threads = gridDim.x * blockDim.x;
    float margin = xlab[1] - xlab[0];
    float margin_inv = 1 / margin;
    for(; n<total_number; n += total_threads)
    {
        // n = idz*MN + idy*M + idx;
        int idx = n % M;
        int idy = (n-idx) % (M*N) / M;
        int idz = (n-idy*M-idx) / (M*N);
       
        k = floor((xvar[n] - xlab[0]) * margin_inv);
        if(k < 0)
        {
            yvar[n] = xvar[n]- xlab[0] + IDX2(ylab, P, D, 0, idz);
        }
        else if(k >= P-1)
        {
            yvar[n] = xvar[n]- xlab[P-1] + IDX2(ylab, P, D, P-1, idz);
        }
        else
        {
            yvar[n] = (IDX2(ylab, P, D, k+1, idz) - IDX2(ylab, P, D, k, idz)) * (xvar[n] - xlab[k]) * margin_inv + IDX2(ylab, P, D, k, idz);
        }
    }
}

/**/
void __global__ nngetp_ker(
        const float *xlab,
        const float *xvar, float *pind,
        int M, int N, int D)
{
    int total_number = M*N*D;
    int n = blockDim.x * blockIdx.x + threadIdx.x;
    int total_threads = gridDim.x * blockDim.x;
    int idx, idy, idz;
    float margin = xlab[1] - xlab[0];
    float margin_inv = 1 / margin;
    for(; n<total_number; n += total_threads)
    {
        idx = n % M;
        idy = (n-idx) % (M*N) / M;
        idz = (n-idy*M-idx) / (M*N);
        
        IDX3(pind, M, N, D, idx, idy, idz) = floor((xvar[n] - xlab[0]) * margin_inv);
    }
}

/* nnback_ker back propagation computing gradients */
void __global__ nnbackx_ker(
        const float *xlab, const float *ylab,
        const float *xvar, const float *yvar,
        float *grad, int M, int N, int D, int P)
{
    int k, l, p, q;
    
    int total_number = M * N * D;
    int n = blockDim.x * blockIdx.x + threadIdx.x;
    int total_threads = gridDim.x * blockDim.x;
    float margin = xlab[1] - xlab[0];
    float margin_inv = 1 / margin;
    for(; n<total_number; n += total_threads)
    {
        // n = idz*MN + idy*M + idx;
        int idx = n % M;
        int idy = (n-idx) % (M*N) / M;
        int idz = (n-idy*M-idx) / (M*N);
     
        k = floor((xvar[n] - xlab[0]) / margin);
        if(k<0 || k>=P-1)
        {
            grad[n] = 1 * yvar[n];
        }
        else
        {
            grad[n] = ((IDX2(ylab, P, D, k+1, idz) - IDX2(ylab, P, D, k, idz)) * margin_inv) * yvar[n];
        }
    }
}

void __global__ nnbackw_ker(
        const float *xlab, const float *ylab,
        const float *xvar, const float *yvar, const float *pind,
        float *grad, int M, int N, int D, int P)
{
  //  __shared__ float INDP[128][128];
  //  __shared__ float L[41];
  //  __shared__ float Y[128][128];
  //  __shared__ float X[128][128];
    
    int m, n, p, q;
    int total_number = D * P;
    int k = blockDim.x * blockIdx.x + threadIdx.x;
    int total_threads = gridDim.x * blockDim.x;
    float margin = xlab[1] - xlab[0];
    float margin_inv = 1 / (margin);
    
    // load global memory to shared memory
    
    
    
    // do computation
    for(; k<total_number; k+=total_threads)
    {
        int idp = k % P;
        int idd = (k-idp) / P;
        float sum = 0;
        for(m=0; m<M; m++)
        {
            for(n=0; n<N; n++)
            {   
                //float temp = ;
                p = (IDX3(pind, M, N, D, m, n, idd));//floor((temp - xlab[0]) / margin);
                //if(p>=0 && p<P-1)
                //{
                    if(p == idp-1 && p>=0 ) //&& p<P-1
                    {
                        // IDX2(grad, P, D, idp, idk) += (1-(IDX3(xvar, M, N, D, m, n, k) - xlab[p]) / margin) * IDX3(yvar, M, N, D, m, n, k);
                        // IDX2(grad, P, D, idp+1, idk) += (IDX3(xvar, M, N, D, m, n, k) - xlab[p]) / margin * IDX3(yvar, M, N, D, m, n, k);
                        sum += (IDX3(xvar, M, N, D, m, n, idd)- xlab[p]) * margin_inv * IDX3(yvar, M, N, D, m, n, idd);
                    }
                    else if(p == idp && p<P-1)
                    {
                        sum += (1 - (IDX3(xvar, M, N, D, m, n, idd) - xlab[p]) * margin_inv) * IDX3(yvar, M, N, D, m, n, idd);
                    }
                //}
                 
            }
        }
        IDX2(grad, P, D, idp, idd) = sum;
    } 
}

void __global__ nnbackw_ker2(
        const float *xlab, const float *ylab,
        const float *xvar, const float *yvar, const float *pind,
        float *grad, int M, int N, int D, int P)
{
    //__shared__ float INDP[128*128];
    __shared__ float L[41];
    //__shared__ float Y[128*128];
    //__shared__ float X[128*128];
    
    int m, n, p, q;
    int total_number = D * P;
    int k = blockDim.x * blockIdx.x + threadIdx.x;
    int total_threads = gridDim.x * blockDim.x;
    float margin = xlab[1] - xlab[0];
    float margin_inv = 1 / (margin);
    
    // load global memory to shared memory
    int idd = blockIdx.x; // t-th channel
    /*for(int t = threadIdx.x; t < M * N; t += blockDim.x)
    {  
       INDP[t] = pind[idd * M * N + t];
    }
    __syncthreads();*/
    
    for(int t = threadIdx.x; t < P; t += blockDim.x)
    {
       L[t] = 0;
    }
    __syncthreads();
    
    // do computation
    for(int t = threadIdx.x; t < M * N; t += blockDim.x)
    {  
       m = t % M;
       n = (t - m) / M; 
       p = pind[idd * M * N + t];
       if(p>=0 && p<P-1)
       {
            float t1 = IDX3(xvar, M, N, D, m, n, idd);
            float t2 = IDX3(yvar, M, N, D, m, n, idd);
            
            L[p] += 1; //(1 - (t1 - xlab[p]) * margin_inv) * t2;
            L[p+1] += 1; //(t1 - xlab[p]) * margin_inv * t2;
       }
    }
    __syncthreads();
    
    for(int t = threadIdx.x; t < P; t += blockDim.x)
    {
       IDX2(grad, P, D, t, idd) = L[t];
    }
    
    /*
    for(; k<total_number; k+=total_threads)
    {
        int idp = k % P;
        int idd = (k-idp) / P;
        float sum = 0;
        for(m=0; m<M; m++)
        {
            for(n=0; n<N; n++)
            {   
                //float temp = ;
                p = INDP; //(IDX3(pind, M, N, D, m, n, idd));//floor((temp - xlab[0]) / margin);
                //if(p>=0 && p<P-1)
                //{
                    if(p == idp-1 && p>=0 ) //&& p<P-1
                    {
                        // IDX2(grad, P, D, idp, idk) += (1-(IDX3(xvar, M, N, D, m, n, k) - xlab[p]) / margin) * IDX3(yvar, M, N, D, m, n, k);
                        // IDX2(grad, P, D, idp+1, idk) += (IDX3(xvar, M, N, D, m, n, k) - xlab[p]) / margin * IDX3(yvar, M, N, D, m, n, k);
                        sum += (IDX3(xvar, M, N, D, m, n, idd)- xlab[p]) * margin_inv * IDX3(yvar, M, N, D, m, n, idd);
                    }
                    else if(p == idp && p<P-1)
                    {
                        sum += (1 - (IDX3(xvar, M, N, D, m, n, idd) - xlab[p]) * margin_inv) * IDX3(yvar, M, N, D, m, n, idd);
                    }
                //}
                 
            }
        }
        IDX2(grad, P, D, idp, idd) = sum;
    } */
}


void mexFunction(int nlhs, mxArray *plhs[], int nrhs, mxArray const *prhs[])
{
    /**/
    mxGPUArray const *xlab;
    mxGPUArray const *ylab;
    mxGPUArray const *xvar;
    float const *d_xlab;
    float const *d_ylab;
    float const *d_xvar;
    int M, N, D, P;
    double margin;
    int i, j, k, l, m, n;
    char const * const errId = "parallel:gpu:mexGPUExample:InvalidInput";
    char const * const errMsg = "Invalid input to MEX file.";
    
    /**/
    int const threadsPerBlock = 256;
    int blocksPerGrid;
    
    /* Initialize the MathWorks GPU API. */
    mxInitGPU();
    
    /**/
    xlab = mxGPUCreateFromMxArray(prhs[0]);
    ylab = mxGPUCreateFromMxArray(prhs[1]);
    xvar = mxGPUCreateFromMxArray(prhs[2]);
    if(mxGPUGetClassID(xlab) != mxSINGLE_CLASS || mxGPUGetClassID(ylab) != mxSINGLE_CLASS || mxGPUGetClassID(xvar) != mxSINGLE_CLASS)
    {
        mexErrMsgIdAndTxt(errId, errMsg);
    }
    d_xlab = (const float *)(mxGPUGetDataReadOnly(xlab));
    d_ylab = (const float *)(mxGPUGetDataReadOnly(ylab));
    d_xvar = (const float *)(mxGPUGetDataReadOnly(xvar));
    
    /* get dimensions */
    const mwSize *xlabdim = mxGPUGetDimensions(xlab);
    const mwSize *ylabdim = mxGPUGetDimensions(ylab);
    const mwSize *xvardim = mxGPUGetDimensions(xvar);
    M = xvardim[0];
    N = xvardim[1];
    D = xvardim[2];
    P = ylabdim[0];
    if(nrhs == 3 && mxIsGPUArray(prhs[0]))
    {
        mxGPUArray *yvar;
        float *d_yvar;
        yvar = mxGPUCreateGPUArray(3, xvardim, mxSINGLE_CLASS, mxREAL, MX_GPU_INITIALIZE_VALUES);
        d_yvar = (float *)(mxGPUGetData(yvar));
        
        /**/
        blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;
        nnline_ker<<<blocksPerGrid, threadsPerBlock>>>(d_xlab, d_ylab, d_xvar, d_yvar, M, N, D, P);
        plhs[0] = mxGPUCreateMxArrayOnGPU(yvar);
        
        mxGPUDestroyGPUArray(yvar);
    }
    else if(nrhs ==4 && mxIsGPUArray(prhs[0]))
    {
        mxGPUArray const *yvar;
        float const *d_yvar;
        mxGPUArray *xgra;
        mxGPUArray *ygra;
        float *d_xgra;
        float *d_ygra;
        yvar = mxGPUCreateFromMxArray(prhs[3]);
        d_yvar = (const float *)(mxGPUGetDataReadOnly(yvar));
        xgra = mxGPUCreateGPUArray(3, xvardim, mxSINGLE_CLASS, mxREAL, MX_GPU_INITIALIZE_VALUES);
        ygra = mxGPUCreateGPUArray(2, ylabdim, mxSINGLE_CLASS, mxREAL, MX_GPU_INITIALIZE_VALUES);
        d_xgra = (float *)(mxGPUGetData(xgra));
        d_ygra = (float *)(mxGPUGetData(ygra));
        
        /**/
        blocksPerGrid = (N * M * D + threadsPerBlock - 1) / threadsPerBlock;
        nnbackx_ker<<<blocksPerGrid, threadsPerBlock>>>(d_xlab, d_ylab, d_xvar, d_yvar, d_xgra, M, N, D, P);
        mxGPUArray *pind;
        pind = mxGPUCreateGPUArray(3, xvardim, mxSINGLE_CLASS, mxREAL, MX_GPU_INITIALIZE_VALUES);
        float *d_pind;
        d_pind = (float *)(mxGPUGetData(pind));
        nngetp_ker<<<blocksPerGrid, threadsPerBlock>>>(d_xlab, d_xvar, d_pind, M, N, D);
        
        int threadsPerBlock2 = threadsPerBlock;
        blocksPerGrid = (D * P + threadsPerBlock2 - 1) / threadsPerBlock2;
        nnbackw_ker<<<blocksPerGrid, threadsPerBlock2>>>(d_xlab, d_ylab, d_xvar, d_yvar, d_pind, d_ygra, M, N, D, P);
        
        
        plhs[0] = mxGPUCreateMxArrayOnGPU(xgra);
        plhs[1] = mxGPUCreateMxArrayOnGPU(ygra);
        mxGPUDestroyGPUArray(xgra);
        mxGPUDestroyGPUArray(ygra);
        mxGPUDestroyGPUArray(yvar);
        mxGPUDestroyGPUArray(pind);
    }
    else
    {
        mexErrMsgIdAndTxt(errId, errMsg);
    }
    
    mxGPUDestroyGPUArray(xlab);
    mxGPUDestroyGPUArray(ylab);
    mxGPUDestroyGPUArray(xvar);
    
}
















